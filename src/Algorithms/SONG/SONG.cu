#include "hip/hip_runtime.h"
/*
* Copyright (C) 2024 by the INTELLI team
 * Created by: Ziao Wang
 * Created on: 2024/11/18
 * Description: [Provide description here]
 */


// #include <Utils/UtilityFunctions.h>

#include <Algorithms/SONG/SONG.hpp>
#include <Utils/IntelliLog.hpp>
#include <ctime>
#include <chrono>
#include <cassert>
#include <vector>
#include <algorithm>
#include <utility>

bool CANDY_ALGO::SONG::setConfig(INTELLI::ConfigMapPtr cfg) {
//  ANNSBase::setConfig(cfg);
  std::string metricType = cfg->tryString("metricType", "L2", true);
  faissMetric = metricType == "L2" ? METRIC_L2 : METRIC_INNER_PRODUCT;
  vecDim = cfg->tryI64("vecDim", 768, true);
  vecVolume = cfg->tryI64("vecVolume", 1000, true);
  data = std::make_unique<SONG_KERNEL::Data> (vecVolume,vecDim);
  if (metricType == "L2") {
    graph = std::make_unique<SONG_KERNEL::KernelFixedDegreeGraph<0>> (data.get());
  } else if (metricType == "IP") {
    graph = std::make_unique<SONG_KERNEL::KernelFixedDegreeGraph<1>> (data.get());
  } else if (metricType == "cos") {
    graph = std::make_unique<SONG_KERNEL::KernelFixedDegreeGraph<2>> (data.get());
  } else {
    INTELLI_WARNING("Switch to L2");
    graph = std::make_unique<SONG_KERNEL::KernelFixedDegreeGraph<0>> (data.get());
  }
  // hipDeviceSetLimit(hipLimitMallocHeapSize,800*1024*1024);
  return true;
}

bool CANDY_ALGO::SONG::insertTensor(const torch::Tensor &t) {
  std::vector<std::vector<std::pair<int,SONG_KERNEL::value_t>>> vertexs;
  INTELLI_INFO("START CONVERT TENSOR TO VECTOR");
  convertTensorToVectorPairBatch(const_cast<torch::Tensor&>(t), vertexs);
  INTELLI_INFO("END CONVERT TENSOR TO VECTOR");
  INTELLI_INFO("START INSERT VERTEX");
  for (auto v : vertexs) {
    data->add(idx,v);
    graph->add_vertex(idx, v);
    // graph->add_vertex_new(idx, v);
    idx++;
  }
  INTELLI_INFO("END INSERT VERTEX");
  return true;
}

bool CANDY_ALGO::SONG::deleteTensor(torch::Tensor &t, int64_t k) {
  return false;
}

bool CANDY_ALGO::SONG::reviseTensor(torch::Tensor &t, torch::Tensor &w) {
  if (t.size(0) > w.size(0) || t.size(1) != w.size(1)) {
    return false;
  }
  // auto idx = findTopKClosest(t, 1, DCOBatchSize);
  // int64_t rows = t.size(0);
  // for (int64_t i = 0; i < rows; i++) {
  //   //auto rowI = t.slice(0, i, i + 1).contiguous();
  //   if (0 <= idx[i]) {
  //     auto rowW = w.slice(0, i, i + 1);
  //     dmBuffer.reviseTensor(idx[i], rowW);
  //     //INTELLI::IntelliTensorOP::editRows(&dbTensor, &rowW, (int64_t) idx);
  //   }
  // }
  return false;
}

std::vector<torch::Tensor> CANDY_ALGO::SONG::searchTensor(const torch::Tensor &q, int64_t k) {
  int query_size = q.size(0);
  std::vector<std::vector<std::pair<int,SONG_KERNEL::value_t>>> queries(query_size);
  std::vector<std::vector<SONG_KERNEL::idx_t>> result_id(query_size);
  convertTensorToVectorPairBatch(const_cast<torch::Tensor&>(q), queries);
  graph->search_top_k_batch(queries, k, result_id);
  std::vector<torch::Tensor> result_t(query_size);

  for (int i = 0; i < query_size; i++) {
    result_t[i] = torch::from_blob(result_id[i].data(), {k}, torch::kInt64).clone();
  }
  return result_t;
}

void CANDY_ALGO::SONG::convertTensorToVectorPair(torch::Tensor &t,
                                      std::vector<std::pair<int,SONG_KERNEL::value_t>> &res) {
    int64_t cols = t.size(1);
    res.resize(cols);
    for(int64_t i = 0; i < cols; i++) {
        res[i] = std::make_pair(i, t[0][i].item<SONG_KERNEL::value_t>());
    }
}

void CANDY_ALGO::SONG::convertTensorToVectorPairBatch(torch::Tensor &ts,
                                    std::vector<std::vector<std::pair<int,SONG_KERNEL::value_t>>> &res) {
    int64_t rows = ts.size(0);
    if (res.size() != rows) {
        res.resize(rows);
    }
    for(int64_t i = 0; i < rows; i++) {
        auto row = ts.slice(0, i, i + 1);
        convertTensorToVectorPair(row, res[i]);
    }
}

bool CANDY_ALGO::SONG::resetIndexStatistics() {
  gpuComputingUs = 0;
  gpuCommunicationUs = 0;
  return true;
}

INTELLI::ConfigMapPtr CANDY_ALGO::SONG::getIndexStatistics() {
  auto cfg = ANNSBase::getIndexStatistics();
  // cfg->edit("hasExtraStatistics", (int64_t) 1);
  // /**
  //  * @brief count of memory access
  //  */
  // cfg->edit("totalMemReadCnt", (int64_t) dmBuffer.getMemoryReadCntTotal());
  // cfg->edit("missMemReadCnt", (int64_t) dmBuffer.getMemoryReadCntMiss());
  // double memMissHitRead = dmBuffer.getMemoryReadCntMiss();
  // memMissHitRead = memMissHitRead / dmBuffer.getMemoryReadCntTotal();
  // cfg->edit("memMissRead", (double) memMissHitRead);
  // cfg->edit("totalMemWriteCnt", (int64_t) dmBuffer.getMemoryWriteCntTotal());
  // cfg->edit("missMemWriteCnt", (int64_t) dmBuffer.getMemoryWriteCntMiss());
  // double memMissHitWrite = dmBuffer.getMemoryWriteCntMiss();
  // memMissHitWrite = memMissHitWrite / dmBuffer.getMemoryWriteCntTotal();
  // cfg->edit("memMissWrite", (double) memMissHitWrite);
  // /**
  //  * @brief gpu statistics
  //  */
  // if (cudaDevice > -1 && torch::cuda::is_available()) {
  //   cfg->edit("gpuCommunicationUs", (int64_t) gpuCommunicationUs);
  //   cfg->edit("gpuComputingUs", (int64_t) gpuComputingUs);
  // } else {
  //   cfg->edit("cpuComputingUs", (int64_t) gpuComputingUs);
  // }
  return cfg;
}
